#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <cfloat>

#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

#include "Ray.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "vec3.h"

typedef unsigned int uint;
typedef unsigned char uchar;

//Sphere X velocity
__device__ static float sphereX = 1.1;
//Sphere Y velocity
__device__ static float sphereY = 0.11;
//Sphere Z velocity
__device__ static float sphereZ = 0.0;

__device__ static float stepSizeX = 0.01;
__device__ static float stepSizeY = 0.01;
__device__ static float stepSizeZ = 0.00;

//Sphere X velocity
__device__ static float sphere2X = 1.1;
//Sphere Y velocity
__device__ static float sphere2Y = 0.11;
//Sphere Z velocity
__device__ static float sphere2Z = 0.0;

__device__ static float stepSize2X = -0.01;
__device__ static float stepSize2Y = -0.01;
__device__ static float stepSize2Z = -0.00;

__device__ vec3 castRay(const ray &r, hitable **world);
__global__ void create_world(hitable **d_list, hitable **d_world);
//#include "bicubicTexture_kernel.cuh"

hipArray *d_imageArray = 0;

extern "C" void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_imageArray;
}
extern "C" void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_imageArray));
}

__global__ void
d_render(uchar4 *d_output, uint width, uint height, hitable **d_world)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint i = y * width + x;
    float u = x / (float)width; //----> [0, 1]x[0, 1]
    float v = y / (float)height;
    u = 2.0 * u - 1.0; //---> [-1, 1]x[-1, 1]
    v = -(2.0 * v - 1.0);
    u *= width / (float)height;
    u *= 2.0;
    v *= 2.0;
    vec3 eye = vec3(0, 0.5, 1.5);
    float distFrEye2Img = 1.0;
    ;
    if ((x < width) && (y < height))
    {
        //for each pixel
        vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);
        //fire a ray:
        ray r;
        r.O = eye;
        r.Dir = pixelPos - eye; //view direction along negtive z-axis!
        vec3 col = castRay(r, d_world);
        float red = col.x();
        float green = col.y();
        float blue = col.z();
        d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
    }
}

// render image using CUDA
extern "C" void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 *output)
{
    /*d_render << <gridSize, blockSize >> > (output, width, height);
     // call CUDA kernel, writing results to PBO memory
     getLastCudaError("kernel failed");*/

    // make our world of hitables
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    create_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    d_render<<<gridSize, blockSize>>>(output, width, height, d_world);
    getLastCudaError("kernel failed");
}

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}
__device__ vec3 castRay(const ray &r, hitable **world)
{
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec))
    {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f,
                           rec.normal.z() + 1.0f);
    }
    else
    {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}
__global__ void create_world(hitable **d_list, hitable **d_world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {

        //Sphere
        //Static Ball*(d_list) = new sphere(vec3(0, 0, -1), 0.5);

        sphereX += stepSizeX;
        sphereY += stepSizeY;
        sphereZ += stepSizeZ;
        if (sphereX > 1.8)
        {
            stepSizeX = -0.01;
        }
        if (sphereY > 1.8)
        {
            stepSizeY = -0.01;
        }
        if (sphereZ > 1.8)
        {
            stepSizeZ = -0.01;
        }

        if (sphereX < -1.8)
        {
            stepSizeX = 0.01;
        }
        if (sphereY < -1.8)
        {
            stepSizeY = 0.01;
        }
        if (sphereZ < -1.8)
        {
            stepSizeZ = 0.01;
        }

        sphere2X += stepSize2X;
        sphere2Y += stepSize2Y;
        sphere2Z += stepSize2Z;
        if (sphere2X > 1.8)
        {
            stepSize2X = -0.01;
        }
        if (sphere2Y > 1.8)
        {
            stepSize2Y = -0.01;
        }
        if (sphere2Z > 1.8)
        {
            stepSize2Z = -0.01;
        }

        if (sphere2X < -1.8)
        {
            stepSize2X = 0.01;
        }
        if (sphere2Y < -1.8)
        {
            stepSize2Y = 0.01;
        }
        if (sphere2Z < -1.8)
        {
            stepSize2Z = 0.01;
        }

        *(d_list) = new sphere(vec3(sphereX, sphereY, sphereZ), 0.2);

        *(d_list + 1) = new sphere(vec3(sphere2X, sphere2Y, sphere2Z), 0.2);

        //Left Wall
        *(d_list + 2) = new sphere(vec3(-10002.0, 0, -3), 10000);
        //Right Wall
        *(d_list + 3) = new sphere(vec3(10002.0, 0, -3), 10000);
        //Top Wall
        *(d_list + 4) = new sphere(vec3(0, 10002.0, -3), 10000);
        //Bottom Wall
        *(d_list + 5) = new sphere(vec3(0, -10002.0, -3), 10000);
        //Back Wall
        *(d_list + 6) = new sphere(vec3(0, 0, -10002.0), 10000);

        *d_world = new hitable_list(d_list, 7);
    }
}
__global__ void free_world(hitable **d_list, hitable **d_world)
{
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
}

#endif
